#include "hip/hip_runtime.h"
/** -*- c++ -*-
 * @file
 * @brief NetVM class - extend TensorVM class, Neural Network Vritual Machine implementation
 *
 * <pre>Copyright (C) 2022- GreenII, this file is distributed under BSD 3-Clause License.</pre>
 */
#include "netvm.h"

#if (T4_DO_OBJ && T4_DO_NN)

__GPU__ void
NetVM::predict(Tensor &I, Tensor &P) {}

///===================================================================
/// private methods
///
__GPU__ int
NetVM::_nnop(t4_layer op) {     /// vtable dispatcher
    VOP(LAYER_OP);
    auto ok = [this,op]() { VLOG(" } %s\n", _op[op]); return 0; };
    ///
    /// handle tensor ops (destructive)
    ///
    VLOG("netvm#nnop %s {", _op[op]);
    if (TOS1T) {
        Tensor &t = TTOS;
        VLOG(" T%d", t.rank);
        switch (op) {
        case L_FLATTEN: t.reshape(t.numel);    return ok();
        case L_RELU:    t.map(RELU);           return ok();
        case L_TANH:    t.map(TANH);           return ok();
        case L_SIGMOID: t.map(SIGM);           return ok();
        case L_SOFTMAX:
            t.map(MUL, RCP(t.sum() + DU_EPS)); return ok();
        case L_LOGSMAX:
            DU sum = t.sum();
            if (sum > DU_EPS) t -= LOG(sum);
            else ERROR("logsoftmax tensor sum < 0!");
            return ok();
        }
        // * continue to zero param
    }
    ///
    /// zero parameter layers
    ///
    if (IS_M(tos)) {
        Model &m = MTOS;
        VLOG(" N%ld", m.numel);
        switch (op) {
        case L_FLATTEN:
        case L_RELU:
        case L_TANH:
        case L_SIGMOID:
        case L_SELU:    m.add(op);           return ok();
        case L_LEAKYRL: m.add(op, 0, 0.01);  return ok();
        case L_ELU:     m.add(op, 0, DU1);   return ok();
        case L_SOFTMAX:
        case L_LOGSMAX: m.add(op);           return ok();
        case L_BATCHNM: m.add(op, 0, 0.1);   return ok(); /// * default momentum=0.1
        }
        // * continue to one param
    }
    ///
    /// one parameter layers
    ///
    if (M1V) {
        DU    a  = POP();
        Model &m = MTOS;
        VLOG(" N%ld %g", m.numel, a);
        switch (op) {
        case L_LINEAR:  m.add(op, INT(a), DU1);        return ok(); /* bias = 1.0 */
        case L_LEAKYRL:
        case L_ELU:     
        case L_DROPOUT: m.add(op, 0, a);               return ok();
        case L_AVGPOOL:
        case L_MAXPOOL: 
        case L_MINPOOL: m.add(op, INT(a));             return ok();
        case L_BATCHNM: m.add(op, 0, a);               return ok();
        case L_USAMPLE: m.add(op, INT(a), UP_NEAREST); return ok();
        }
        PUSH(a);                                   /// * restore tos
        /// continue to error handling cases
    }
    switch (op) {
    case L_LINEAR:
        if (M2V) {                                 /// * param checking
            U32 c    = POPi;                       ///> number of output channels
            DU  bias = POP();                      ///> bias range [-bias, bias)
            VLOG(" N%ld c=%d bias=%g", MTOS.numel, c, bias);
            MTOS.add(op, c, bias);                 /// * (N b c -- N')
        }
        else ERROR("( N [bias] n -- ) for linear required!");
        break;
    case L_FLATTEN:
    case L_SELU:
    case L_SOFTMAX:
    case L_LOGSMAX: ERROR("( N -- ) no param needed!"); break;
    case L_LEAKYRL:
    case L_ELU:
    case L_DROPOUT:
    case L_AVGPOOL:
    case L_MAXPOOL: 
    case L_MINPOOL:
    case L_BATCHNM: ERROR("( N n -- ) one param required!"); break;
    case L_USAMPLE:
        if (M2V) {
            U16 n = POPi;
            DU  m = POP();
            VLOG(" N%ld n=%d m=%g", MTOS.numel, n, m);
            MTOS.add(op, n, m);
        }
        else ERROR("( N [mtum] n -- ) for upsample required?");
        break;
    default:
        if (!IS_OBJ(tos)) {
            switch (op) {
            case L_RELU:    xop1(RELU, DU0); break;
            case L_TANH:    xop1(TANH);      break;
            case L_SIGMOID: xop1(SIGM);      break;
            }
        }
        else ERROR("layer %d not supported(2)\n", op);
    }
    return ok();
}
///
/// dataset ops
///
__GPU__ void
NetVM::_pickle(bool save) {                 ///< ( N addr len -- ) or ( N addr len mode -- )
    U8   mode= save ? FAM_WO : FAM_RW;      ///< file access mode

    if (ss.idx > 1 && IS_OBJ(ss[-2])) { /* OK */ }
    else if (ss.idx > 2 && IS_OBJ(ss[-3])) mode |= POPi;       ///< TODO: RAW format
    else { ERROR("(model|tensor) adr len [mode]?\n"); return; }
    
    IU   len = POPi;                        ///< string length (not used for now)
    IU   adr = POPi;                        ///< address to pmem
    char *fn = (char*)mmu.pmem(adr);        ///< pointer to string on PAD
    sys.op(IS_M(tos) ? (save ? OP_NSAVE : OP_NLOAD) : OP_TSAVE, mode, tos);
    state = HOLD;                           /// * return to CPU
}

///
/// fetch parameters onto TOS
/// n=0:W, 1:B, 2:dW, 3:dB
///
__GPU__ void
NetVM::_get_parm(int n) {
    if (!M1V) { ERROR("N n required?"); return; }
    
    S16 i = POPi;
    Tensor *p = MTOS[i].grad[n];
    if (p) {
        DU v = mmu.obj2du(*p);
        PUSH(DUP(v));
    }
    else PUSH(DU0);
}
///
/// fetch parameters onto TOS
/// n=0:W, 1:B, 2:dW, 3:dB
///
__GPU__ void
NetVM::_set_parm(int n) {
    if (!MTV) { ERROR("N T n required?"); return; }

    S16    i  = POPi;
    Tensor &p = *MNOS[i].grad[n];
    Tensor &t = TTOS;
    if (t.numel == p.numel) {
        Tensor::copy(t, p);
        DU t = POP(); DROP(t);
    }
    else {
        PUSH(i);                        /// * restore n
        ERROR("Tensor and model parameter is not the same shape");
    }
}
/// Convolution ops
/// @default: kxk filter, padding=1, stride=1, dilation=1
/// @parameters
///    k: kernel size
///
__GPU__ void
NetVM::_conv(U16 k) {
    U16 opt[] = { k, k, 1, 1, 1 };      ///> default config vector
    if (TOS1T) {                        ///> if optional vector given
        Tensor &v = TTOS;
        if (v.rank == 1) {
            for (int i=0; i<5; i++) opt[i] = (U16)v.data[i];
            DU t = POP(); DROP(t);
        }
        else { ERROR("vec?"); return; }
    }
    if (!M2V) { ERROR("Model#add bias c for conv2d required!"); return; }
    U32 c    = POPi;                    ///> number of output channels
    DU  bias = POP();                   ///> convolution bias
    VLOG("netvm#conv { N%ld k=%d c=%d bias=%g", MTOS.numel, k, c, bias);
    MTOS.add(L_CONV, c, bias, opt);
    VLOG(" } conv\n");
}
///
/// loss functions
///
__GPU__ void
NetVM::_loss(t4_loss op) {
    if (TOS2T) {                        /// * calculate loss of two tensors
        DU y = POP();                   /// * pop off target tensor
        DU n = TTOS.loss(op, (Tensor&)mmu.du2obj(y));
        PUSH(n);
        DROP(y);                        /// * free target tensor
    }
    else if (TOS1T && IS_M(ss[-1])) {   /// * model loss
        DU y = POP();
        DU n = MTOS.loss(op, (Tensor&)mmu.du2obj(y));
        PUSH(n);                        /// * loss on TOS
        DROP(y);                        /// * pop off t
    }
    else if (IS_M(tos)) PUSH(MTOS.loss(op));
    else ERROR("model?\n");
}
///===================================================================
///
/// Neural Network Vocabulary
///
__GPU__ void
NetVM::init() {
    if (id!=0) return;                        /// * singleton
    TensorVM::init();
    ///
    ///@defgroup Model creation and persistence
    ///@{
    CODE("nn.model",                          ///> (n h w c -- N)
         if (ss.idx < 4 ||                    /// * param check
             IS_OBJ(tos) || IS_OBJ(ss[-1]) ||
             IS_OBJ(ss[-2]) || IS_OBJ(ss[-3])) {
             ERROR("n h w c?\n"); return;
         }
         U32 c=POPi; U32 w=POPi; U32 h=POPi; U32 n=POPi;
         Model  &m = mmu.model();             /// * create NN model
         Tensor &t = mmu.tensor(n,h,w,c);     /// * create input tensor
         m.trace(sys.trace());                /// * set model tracing control
         m.npush(t);                          /// * serves as the 1st layer
         PUSH(m));
    ///@}
    ///@defgroup Convolution and Linear ops
    ///@{
    CODE("conv1x1",   _conv(1));              ///> (N b c -- N')
    CODE("conv2d",    _conv(3));              ///> (N b c [A] -- N')
    CODE("linear",    _nnop(L_LINEAR));       ///> (N b c -- N')
    ///@}
    ///@defgroup BatchNorm and Activation ops
    ///@{
    CODE("relu",      _nnop(L_RELU));         ///> (N -- N')
    CODE("tanh",      _nnop(L_TANH));         ///> (N -- N')
    CODE("sigmoid",   _nnop(L_SIGMOID));      ///> (N -- N')
    CODE("selu",      _nnop(L_SELU));         ///> (N -- N')
    CODE("leakyrelu", _nnop(L_LEAKYRL));      ///> (N a -- N')
    CODE("elu",       _nnop(L_ELU));          ///> (N a -- N')
    CODE("softmax",   _nnop(L_SOFTMAX));      ///> (N -- N')
    CODE("logsoftmax",_nnop(L_LOGSMAX));      ///> (N -- N')
    CODE("batchnorm", _nnop(L_BATCHNM));      ///> (N -- N')
    ///@}
    ///@defgroup Pooling, Dropout, and Upsample ops
    ///@{
    CODE("maxpool",   _nnop(L_MAXPOOL));      ///> (N n -- N')
    CODE("avgpool",   _nnop(L_AVGPOOL));      ///> (N n -- N')
    CODE("minpool",   _nnop(L_MINPOOL));      ///> (N n -- N')
    CODE("dropout",   _nnop(L_DROPOUT));      ///> (N p -- N')
    CODE("upsample",  _nnop(L_USAMPLE));      ///> (N [m] n -- N')
    ///@}
    ///@defgroup Loss functions
    ///@{
    CODE("loss.mse",  _loss(LOSS_MSE));       ///> (N T -- N T n) mean square error
    CODE("loss.bce",  _loss(LOSS_BCE));       ///> (N T -- N T n) binary cross-entropy
    CODE("loss.ce",   _loss(LOSS_CE));        ///> (N T -- N T n) cross-entropy
    CODE("loss.nll",  _loss(LOSS_NLL));       ///> (N T -- N T n) negative log-likelihood
    CODE("nn.loss",                           ///> (N T -- N T n) auto select loss function
         if (IS_M(tos) || (TOS1T && IS_M(ss[-1]))) {
             Model &m = IS_M(tos) ? MTOS : (Model&)mmu.du2obj(ss[-1]);
             switch (m[-2].grad_fn) {
             case L_TANH:
             case L_SIGMOID: _loss(LOSS_BCE); break;
             case L_SOFTMAX: _loss(LOSS_CE);  break;
             case L_LOGSMAX: _loss(LOSS_NLL); break;
             default:        _loss(LOSS_MSE);
             }
         }
         else ERROR("TOS is not a tensor or NOS is not a model!\n"));
    ///@}
    ///@defgroup Gradiant ops
    ///@{
    CODE("nn.zero",
         if (IS_M(tos)) MTOS.grad_zero();
         else ERROR("TOS is not a model!\n"));
    CODE("nn.sgd",                            
         if (M2V) {                           ///> (N p m -- N')
             DU m  = POP();                   ///< momentum
             DU lr = POP();                   ///< learn rate
             MTOS.sgd(lr, m);
         }
         else if (M1V) {                      ///> (N p -- N')
             DU lr = POP();                   ///< learn rate
             MTOS.sgd(lr, DU0);               ///< default momentum = 0.0
         }
         else ERROR("rate mtum nn.sgd?\n"));
    CODE("nn.adam",
         if (M1V) {                           ///> (N lr -- N')
             DU lr = POP();                   /// * learing rate 
             MTOS.adam(lr, 0.9, 0.999);       /// * default b1=0.9, b2=0.999
         }
         else if (M2V) {                      ///> (N lr b1 -- N')
             DU b1 = POP();                   ///< beta1 i.g. 0.9
             DU lr = POP();                   ///< learning rate i.g. 0.001
             MTOS.adam(lr, b1, 0.999);
         }
         else ERROR("rate beta1 nn.adam?\n"));
    CODE("nn.onehot",                         /// * current onehot vector
         if (IS_M(tos)) {
             Tensor &hot = MTOS.onehot();
             DU v = mmu.obj2du(hot);
             PUSH(DUP(v));
         }
         else ERROR("TOS is not a model!\n"));
    CODE("nn.hit", 
         if (IS_M(tos)) PUSH(I2D(MTOS.hit()));
         else ERROR("TOS is not a model!\n"));
    ///@}
    ///@defgroup Batch Control ops
    ///@{
    CODE("trainable",
         if (M1V) { bool on = POPi; MTOS.train = on; }
         else ERROR("N [1|0] required\n"));
    CODE("batchsize",
         if (IS_M(tos)) PUSH(MTOS.batch_size());
         else ERROR("TOS is not a model?\n"));
    CODE("dataset",                             /// * create a dataset
         char    *dsn = sys.fetch();            ///< retrieve dataset name
         Dataset &ds  = mmu.dataset(POPi);      ///< batch size
         PUSH(mmu.obj2du((T4Base&)ds));         /// * create a dataset as TOS
         sys.op(OP_DATA, 0, tos);               /// * issue a dataset init
         sys.op_fn(dsn);                        /// * send dataset name
         state = HOLD);
    CODE("fetch",  sys.op(OP_FETCH, 0, tos));   /// * fetch a dataset batch
    CODE("rewind", sys.op(OP_FETCH, 1, tos));   /// * rewind a dataset (batch_id=0)
    CODE("forward",                             /// * forward process
         if (IS_M(ss[-1]) && TOS1D) {           /// * TOS is a dataset
             DU x = POP();                      /// * NOS is the model
             MTOS.forward((Tensor&)mmu.du2obj(x));     /// * exec forward path
             DROP(x);                           /// * release reference
         }
         else if (IS_M(tos) && IS_OBJ(rs[-1])) {       /// * in a for/next loop
             Tensor &t = (Tensor&)mmu.du2obj(rs[-1]);  /// * rs[-1] is a dataset
             if (t.is_dataset()) MTOS.forward(t);
             else ERROR("rs[-1] is not a dataset?\n");
         }
         else ERROR("no model or a dataset?\n"));
    CODE("backprop",
         if (IS_M(ss[-1]) && TOS1T) {                  /// * TOS is a onehot vector
             DU y = POP();                     
             MTOS.backprop((Tensor&)mmu.du2obj(y));    /// * backprop(target vector)
             DROP(y);
         }
         else if (IS_M(tos)) MTOS.backprop();          /// * use default output
         else ERROR("TOS not a model?\n"));
    CODE("broadcast",
         if (IS_M(ss[-1]) && TOS1T) {                  /// * TOS is a onehot vector
             DU y = POP();
             MTOS.broadcast((Tensor&)mmu.du2obj(y));
             DROP(y);
         }
         else ERROR("TOS not a tensor nor NOS a model?\n"));
    ///@}
    ///@defgroup Debugging ops
    ///@{
    CODE(">n",      if (M1V) { DU t = POP(); MTOS.npush(t); });
    CODE("n@",      if (!M1V) return;
         S32    i  = POPi;
         Tensor &t = MTOS[i];
         DU     v  = mmu.obj2du(t);
         PUSH(DUP(v)));
    CODE("nn.w",    _get_parm(0));                 ///< tensor.weight
    CODE("nn.b",    _get_parm(1));                 ///< tensor.bias
    CODE("nn.dw",   _get_parm(2));                 ///< tensor.weight.grad
    CODE("nn.db",   _get_parm(3));                 ///< tensor.bias.grad
    CODE("nn.w=",   _set_parm(0));                 ///< populate tensor.weight
    CODE("nn.b=",   _set_parm(1));                 ///< populate tensor.bias
    CODE("network", if (IS_M(tos)) sys.dot(DOT, tos));  ///< non destructive
    ///
    /// ===========================================================================
    ///
    /// * overwrite/extended word
    ///
    CODE("boot",      mmu.clear(FIND((char*)"network") + 1));
    CODE("flatten",   _nnop(L_FLATTEN));
    CODE("save",      _pickle(true));              /// * save trainned model
    CODE("load",      _pickle(false));             /// * load trainned model
    
    TRACE("NetVM::init ok, sizeof(Model)=%ld\n", sizeof(Model));
};

#endif  // (T4_DO_OBJ && T4_DO_NN)
//===========================================================================
