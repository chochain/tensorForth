#include "hip/hip_runtime.h"
/** -*- c++ -*-
 * @file
 * @brief Tensor class - ranked tensor impmementation i.e. vector, matrix, tensor, ...
 *
 * <pre>Copyright (C) 2022- GreenII, this file is distributed under BSD 3-Clause License.</pre>
 */
#include "tensor.h"

#if T4_DO_OBJ
///=======================================================================
/// static methods
///
/// k_sum1 - sum all elements into one value (used by matrix)
/// Note: use stride adding in parallel, instead of atomicAdd
///
__KERN__ void
k_sum1(DU *I, DU *sum, U64 numel) {                              ///< sum all elements
    DU const z = { d_sum(I, numel) };
    if (threadIdx.x == 0) *sum = z;
}
///
__KERN__ void
k_var1(DU *I, DU avg, DU *var, U64 numel) {
    DU const nv = { d_nvar(I, avg, numel) };
    if (threadIdx.x == 0 && numel) *var = nv / numel;
}

__KERN__ void
k_matmul(
    DU *A, DU *B, DU *O,   /* O[H*W*C] = A[H*K*C] @ B[K*W*C] */
    t4_mm_opt opt,
    U32 K, U32 H, U32 W)
{
    const U32 j  = blockIdx.x * blockDim.x + threadIdx.x;  ///< W  2T  range
    const U32 i  = blockIdx.y * blockDim.y + threadIdx.y;  ///< H  65M range
    const U32 c  = blockIdx.z,  C = gridDim.z;             ///< C
    const U64 z0 = ((U64)W * i + j) * C + c;               ///< output matrix index
    
    if (i < H && j < W && c < C) {                         /// * TODO: tiled
        DU  *ax, *bx;
        U64 ai, bi;
        if (opt & MM_A_TXP) {                              /// * transpose A
            ax = &A[(U64)C * i + c]; ai = (U64)H * C;
            bx = &B[(U64)C * j + c]; bi = (U64)W * C;
        }
        else if (opt & MM_B_TXP) {                         /// * transpose B
            ax = &A[(U64)C * K * i + c]; ai = (U64)C;
            bx = &B[(U64)C * K * j + c]; bi = (U64)C;
        }
        else {                                             /// * no tranposition
            ax = &A[(U64)C * K * i + c]; ai = (U64)C;
            bx = &B[(U64)C * j + c];     bi = (U64)W * C;
        }
        DU2 acc = DU0;                                     /// * TODO: suffle sum
//      acc += ax[k * C] * bx[k * N * C];                  /// * 8.1 ms 1Kx1K
        for (U32 k = 0; k < K; k++, ax += ai, bx += bi) {
            acc += (*ax) * (*bx);                          /// * 6.2 ms 1Kx1K
        }
        if (opt & MM_INC) O[z0] += acc;                    /// * increment O
        else              O[z0] =  acc;                    /// * overwrite O
    }
}
///
/// GEMM kernel (used CUDA dynamic parallelism)
///     O = alpha * A x B + beta * O
///     where A = HxKxC, B = KxWxC, O = HxWxC
///
__KERN__ void
k_gemm(
    DU *A, DU *B, DU *O,  /* O[H*W*C] = a * A[H*K*C] @ B[K*W*C] + b * O[H*W*C] */
    DU alpha, DU beta,
    U32 K, U32 H, U32 W)
{
    const U32 j = threadIdx.x + blockIdx.x * blockDim.x;   ///< W
    const U32 i = threadIdx.y + blockIdx.y * blockDim.y;   ///< H
    const U32 c = blockIdx.z, C = gridDim.z;               ///< channel deep
    const U64 WC= W * C;
    const U64 z0= ((U64)W * i + j) * C + c;                ///< output index

    if (i < H && j < W && c < C) {                         /// * TODO: tiled
        DU *ax = &A[(U64)C * K * i + c];
        DU *bx = &B[(U64)C * j + c];
        DU2 acc = DU0;                                     /// * TODO: suffle sum
        for (U32 k = 0; k < K; k++, ax += C, bx += WC) {
            acc += (*ax) * (*bx);
        }
        O[z0] = alpha * acc + beta * O[z0];                /// * scaling
    }
}
///
/// tensor-scalar addition O = A op n element-wise (Hadamard)
///
__GPU__ Tensor&
Tensor::ten_op(math_op op, Tensor &A, DU v, Tensor &O) {
    U32 N = A.N(), H = A.H(), W = A.W(), C = A.C();
    _OP(MATH_OP);
    MM_DB("  tensor#ten_op O[%d,%d,%d,%d] = A %s %6.2f\n", N, H, W, C, _op[op], v);

    FORK1(k_ts_op, A.numel, op, A.data, v, O.data);
    CDP_SYNC();
    return O;
}
///
/// tensor-tensor element-wise C = A op B where op=ADD|SUB|MUL|DIV (Hadamard)
///
__GPU__ Tensor&
Tensor::ten_op(math_op op, Tensor &A, Tensor &B, Tensor &O) {
    U32 N = A.N(), H = A.H(), W = A.W(), C = A.C();
    _OP(MATH_OP);
    MM_DB("  tensor#ten_op O[%d,%d,%d,%d] = A %s B\n", N, H, W, C, _op[op]);
    
    FORK1(k_tt_op, A.numel, op, A.data, B.data, O.data);
    CDP_SYNC();
    return O;
}
__GPU__ Tensor&
Tensor::sum(Tensor &A, Tensor &O) {
    U32 N = A.N(), H = A.H(), W = A.W(), C = A.C();
    MM_DB("  tensor#sum A[%d,%d,%d,%d] => O[%d, %d]\n", N, H, W, C, N, C);
    O.fill(DU0);
    FORK4(k_nsum, A.data, O.data, (U64)H*W);
    CDP_SYNC();
    return O;
}
__GPU__ Tensor&
Tensor::var(Tensor &A, Tensor &G, Tensor &O) {
    U32 N = A.N(), H = A.H(), W = A.W(), C = A.C();
    MM_DB("  tensor#var A[%d,%d,%d,%d] => O[%d,%d]\n", N, H, W, C, N, C);
    sum(A, G);
    G *= DU1 / (H*W);
    O.fill(DU0);
    FORK4(k_nvar, A.data, G.data, O.data, (U64)H*W);
    CDP_SYNC();
    for (int i=0; i< O.numel; i++) {
        O.data[i] = SQRT(O.data[i] / (H*W));
    }
    return O;
}
__GPU__ Tensor&
Tensor::mm(
    Tensor &A, Tensor &B, Tensor &O, t4_mm_opt opt) {
    U32 H  = opt & MM_A_TXP ? A.W() : A.H();
    U32 Ka = opt & MM_A_TXP ? A.H() : A.W();
    U32 W  = opt & MM_B_TXP ? B.H() : B.W();
    U32 Kb = opt & MM_B_TXP ? B.W() : B.H();
    U32 N  = B.N(), C = B.C();                     /// B, O common dimensions
    if (Ka != Kb || N != O.N() || C != O.C()) {
        ERROR("  tensor#mm Ka(%d)!=Kb(%d) or N, C diff\n", Ka, Kb);
        return O;
    }
    MM_DB("  tensor#matmul K=%d => NHWC=[%d,%d,%d,%d]\n", Ka, N, H, W, C);
    
    for (U32 n = 0; n < N; n++) {
        DU *da = A.data, *db = B.slice(n), *dx = O.slice(n);
        FORK3(k_matmul, H, W, C, da, db, dx, opt, Ka);
    }
    CDP_SYNC();
    return O;
}
///
/// tensor GEMM C' = alpha * A x B + beta * C
///
__GPU__ Tensor&
Tensor::gemm(Tensor &A, Tensor &B, Tensor &O, DU alpha, DU beta) {
    U32 H = A.H(), W = B.W(), Ka = A.W(), Kb = B.H();
    U32 N = B.N(), C = B.C();
    if (Ka != Kb || N != O.N() || C != O.C()) {
        ERROR("  tensor#gemm ka(%d)!=kb(%d) or N, C diff\n", Ka, Kb);
        return O;
    }
    MM_DB("  tensor#gemm K=%d, a=%g, b=%g => NHWC=[%d,%d,%d,%d]\n",
          Ka, alpha, beta, N, H, W, C);

    for (U32 n = 0; n < N; n++) {
        DU *da = A.data, *db = B.slice(n), *dx = O.slice(n);
        FORK3(k_gemm, H, W, C, da, db, dx, alpha, beta, Ka);
    }
    CDP_SYNC();
    return O;
}
__GPU__ Tensor&
Tensor::copy(Tensor &A, Tensor &O) {
    MM_DB("  tensor#copy %p to %p numel=%ld\n", A.data, O.data, A.numel);
    FORK1(k_copy, A.numel, A.data, O.data);
    CDP_SYNC();
    return O;
}
__GPU__ Tensor&
Tensor::transpose(Tensor &A, Tensor &T) {
    U32 N = A.N(), H = A.H(), W = A.W(), C = A.C();
    MM_DB("  tensor#transpose A[%d,%d,%d,%d]\n", N, H, W, C);
    
    for (U32 n = 0; n < N; n++) {
        DU *da = A.slice(n), *dt = T.slice(n);
        FORK3(k_transpose, H, W, C, da, dt);
    }
    CDP_SYNC();
    return T;
}
///
/// matrix inversion (Gauss-Jordan with Pivot)
/// Note: Gauss-Jordan elimination is expensive O(N^3)
/// TODO: CDP
///
__GPU__ Tensor&
Tensor::inverse(Tensor &A, Tensor &I) {
    U32 m = A.H(), n = A.W();
    MM_DB("  tensor#inverse [%d,%d]\n", m, n);
    if (m != n) { ERROR("square matrix?"); return I; }
    DU *da = A.data, *di = I.data;
    auto swap_rows = [da, di, n](U32 u, U32 z) {
        for (U32 k = 0; k < n; k++) {         ///> TODO: swap entire row
            DU ta = da[k + z * n], ti = di[k + z * n];
            da[k + z * n] = da[k + u * n]; da[k + u * n] = ta;
            di[k + z * n] = di[k + u * n]; di[k + u * n] = ti;
        }
    };
    auto find_max = [da, n](U32 z) {
        int u = z;
        for (U32 i = z + 1; i < n; i++) {    ///> TODO: CDP reduce
            if (ABS(da[z + i * n]) > ABS(da[z + u * n])) u = i;
        }
        if (ABS(da[z + u * n]) < DU_EPS) {
            ERROR("tensor#inverse sigular!\n");
            return -1;
        }
        return u;
    };
    auto diag = [da, di, n](U32 z) {
        DU r0 = da[z + z * n];
        for (U32 k = 0; k < n; k++) {
            U32 i = k + z * n;
            di[i] /= r0;
            da[i] /= r0;
        }};
    auto elim = [da, di, n](U32 z) {
        for (U32 i = 0; i < n; i++) {
            DU r1 = da[z + i * n];
            for (U32 k = 0; i!=z && k < n; k++) {
                di[k + i * n] -= r1 * di[k + z * n];
                da[k + i * n] -= r1 * da[k + z * n];
            }
        }};
    for (U32 z = 0; z < n; z++) {
        int u = find_max(z);
        if (u < 0) break;
        else if (u != z) {
            swap_rows(u, z);
        }
        diag(z);
        elim(z);
    }
    return I;
}
///
/// LU decomposition (no Pivot)
/// Note: A stores both L and U in-place to save space
/// TODO: CDP
///
__GPU__ Tensor&
Tensor::lu(Tensor &A) {
    U32 m = A.H(), n = A.W();
    MM_DB("  tensor#lu [%d,%d]\n", m, n);
    if (m != n) { ERROR("square matrix?"); return A; }

    DU *da = A.data;
    auto elim = [da, n](U32 z) {
        DU ra = da[z + z * n];
        if (fabs(ra) < DU_EPS) return;      /// * if 0 skip the row
        for (U32 y = z + 1; y < n; y++) {
            DU r1 = da[z + y * n] / ra;     /// * substitution
            for (U32 k = z; k < n; k++) {
                da[k + y * n] -= r1 * da[k + z * n];
            }
            da[z + y * n] = r1;             /// L stored in A to save space
        }
    };
    for (U32 z = 0; z < n; z++) {
        elim(z);               /// * eliminate variables in upper triangle
	}
    return A;
}
///
/// LU (preprocessed) matrix inversion
/// TODO: CDP
///
__GPU__ Tensor&
Tensor::lu_inverse(Tensor &LU) {
    U32 m = LU.H(), n = LU.W();
    MM_DB("  tensor#lu_inverse [%d,%d]\n", m, n);
    DU *dd = LU.data;
    auto forward = [dd, n](int z) {
        for (int y = z + 1; y < n; y++) {
            DU r1 = dd[z + y * n];
            for (int k = 0; k < z; k++) {               // columns before
                dd[k + y * n] -= dd[k + z * n] * r1;
            }
            dd[z + y * n] = -r1;                        // current z column
        }};
    auto backward = [dd, n](int z) {
        DU r0 = RCP(dd[z + z * n]);
        dd[z + z * n] = r0;                             // diag
        for (int k = z + 1; k < n; k++) {               // current z row
            dd[k + z * n] *= r0;
        }
        for (int y = 0; y < z; y++) {                   // factorize rows above
            DU r1 = dd[z + y * n];
            dd[z + y *  n] = -r1 * r0;                  // current z column
            for (int k = z + 1; k < n; k++) {           // columns after
                dd[k + y * n] -= dd[k + z * n] * r1;
            }
        }};
    
    if (LU.det() < DU_EPS) return LU;
    
    for (int z = 0; z < n - 1; z++)  forward(z);
    for (int z = n - 1; z >= 0; z--) backward(z);
    
    return LU;
}
///
/// PLU methods with permutation vector
/// Note: A stores both L and U in-place to save space, use triu, trul to extract
///       P is permutation vector
/// TODO: CDP
///
__GPU__ Tensor&
Tensor::plu(Tensor &A, Tensor &P, int *ns) {
    U32 m = A.H(), n = A.W();
    MM_DB("  tensor#plu [%d,%d]\n", m, n);
    if (m != n) { ERROR("square matrix?"); return A; }

    DU *da = A.data, *dp = P.data;
    *ns = 0;                                  ///> initialize flip sign
    auto swap_rows = [da, dp, n](U32 u, U32 z) {
        DU t = dp[z]; dp[z] = dp[u]; dp[u] = t;
        for (U32 k = z; k < n; k++) {         ///> TODO: swap entire row
            t = da[k + z * n];
            da[k + z * n] = da[k + u * n];
            da[k + u * n] = t;
        }
    };
    auto find_max = [da, n](U32 z) {
        int u = z;
        for (U32 i = z + 1; i < n; i++) {    ///> TODO: CDP reduce
            if (ABS(da[z + i * n]) > ABS(da[z + u * n])) u = i;
        }
        if (ABS(da[z + u * n]) < DU_EPS) {
            MM_DB("  tensor#lu sigular!\n");
            return -1;
        }
        return u;
    };
    auto elim = [da, n](U32 z) {
        DU ra = da[z + z * n];
        if (fabs(ra) < DU_EPS) return;       /// * if 0 skip the row
        for (U32 y = z + 1; y < n; y++) {
            DU r1 = da[z + y * n] / ra;      /// * substitution
            for (U32 k = z; k < n; k++) {
                da[k + y * n] -= r1 * da[k + z * n];
            }
            da[z + y * n] = r1;              /// L stored in A to save space
        }
    };
    for (U32 z = 0; z < m; z++) dp[z] = z;   /// init permutation vector
    for (U32 z = 0; z < n; z++) {
        int u = find_max(z);   /// * pivot to reduce rounding error
        if (u < 0) return A;
        if (u != z) {          /// * swapping row which has maximum xth column element
            swap_rows(u, z);
            *ns += 1;
        }
        elim(z);               /// * eliminate variables in upper triangle
    }
    return A;
}

///=======================================================================
/// tensor arithmetics
///
__GPU__ DU
Tensor::sum() {
    static DU z;                                    ///< shared static memory
    FORK1(k_sum1, numel, data, &z);
    CDP_SYNC();
    return SCALAR(z);
}
__GPU__ DU
Tensor::avg() {
    DU v = sum() / numel;
    return SCALAR(v);
}
__GPU__ DU
Tensor::std() {
    static DU var;
    FORK1(k_var1, numel, data, avg(), &var);     /// * 8x straight loop
    CDP_SYNC();

    DU v = numel ? SQRT(var) : DU0;
    return SCALAR(v);
}
__GPU__ DU
Tensor::max() {
    DU v = data[0];
    for (U64 i=1; i < numel; i++) {              ///> TODO: CDP prefix sum
        v = MAX(data[i], v);
    }
    return SCALAR(v);
}
__GPU__ DU
Tensor::min() {
    DU v = data[0];
    for (U64 i=1; i < numel; i++) {              ///> TODO: CDP prefix sum
        v = MIN(data[i], v);
    }
    return SCALAR(v);
}
__GPU__ DU
Tensor::dot(Tensor &B) {
    DU  acc = DU0;
    if (rank == 1 && B.rank == 1 && numel == B.numel) {
        for (U64 k=0; k < numel; k++) {          ///> TODO: kernel
            acc += data[k] * B.data[k];
        }
    }
    else ERROR("A.dot(B) dim? %ld != %ld)\n", numel, B.numel);
    return SCALAR(acc);
}
__GPU__ DU
Tensor::loss(t4_loss op, Tensor &tgt) {
    /*
    auto check_bce = [this, &tgt]() {
        DU sum = DU0;
        for (int i=0; i<numel; i++) {
            DU t = tgt.data[i], y = this->data[i];
            sum += t * LN(y + DU_EPS) + (DU1-t) * LN(DU1 - y + DU_EPS);
        }
        return -sum;
    };
    */
    DU z = DU0;                      ///> result loss value
    switch (op) {
    case LOSS_MSE:                   /// * mean squared error, input from linear
        *this -= tgt;                /// * (output - predict)
        *this *= *this;              /// * (output - predict)^2
        z = 0.5 * sum();
        break;
    case LOSS_BCE: {                 /// * binary cross_entropy, input from sigmoid
        FORK1(k_bce, numel, data, tgt.data);
        CDP_SYNC();
        z = -sum();                  /// * -(y * ln(out_i) + (1-y) * ln(1-out_i))
    } break;
    case LOSS_CE:                    /// * cross_entropy, input from softmax
        map(LN);                     /// * log(out_i)
        /* no break */
    case LOSS_NLL:                   /// * negative log likelihood, input from log-softmax
        *this *= tgt;                /// * out_i * tgt_i
        z = -sum();                  /// * sum for mini-batch samples
        break;
    default: ERROR("Model#loss op=%d not supported!\n", op);
    }
    z /= N();                        /// * mini-batch average
    
    return SCALAR(z);                /// make sum a scalar value (not object)
}
///=======================================================================
/// linear algebra methods
///=======================================================================
/// matrix determinant
///
__GPU__ DU
Tensor::det() {
    U32 m = H(), n = W();
    MM_DB("  tensor#det [%d,%d]\n", m, n);

    DU v = DU1;
    for (U32 z = 0; z < m; z++) v *= data[z + z * n];

    return SCALAR(v);
}
///
/// matrix upper triangle
///
__GPU__ Tensor&
Tensor::triu() {
    U32 m = H(), n = W();
    MM_DB("  tensor#upper [%d,%d]\n", m, n);

    for (U32 z = 1; z < m; z++) {
        for (U32 k = 0; k < z; k++) {
            data[k + z * n] = DU0;
        }
    }
    return *this;
}
///
/// matrix lower triangle with diag filled with 1
///
__GPU__ Tensor&
Tensor::tril() {
    U32 m = H(), n = W();
    MM_DB("  tensor#lower [%d,%d]\n", m, n);

    for (U32 z = 0; z < m; z++) {
        data[z + z * n] = DU1;
        for (U32 k = z + 1; k < n; k++) {
            data[k + z * n] = DU0;
        }
    }
    return *this;
}
///=======================================================================
/// Tensor life-cycle ops
///
__BOTH__ Tensor&
Tensor::reset(void *mem, U64 sz, t4_obj tt, t4_layer fn) {
    MM_DB("  tensor#reset(%p,%ld)\n", mem, sz);
    init(sz, tt, 1);                                   /// T4Base attributes

    const U64 GB   = 1L << 30;
    const U16 s[4] = { 1, 1, 1, 1 };
    const U32 h[4] = {
        (U32)(sz > GB ? (sz>>30) : sz),
        (U32)(sz > GB ? GB : 1L),
        1, 1
    };
    const Tensor *t[4]= { NULL, NULL, NULL, NULL };
    data    = (DU*)mem;
    grad_fn = fn;
    memcpy(stride, s, sizeof(s));
    memcpy(shape,  h, sizeof(h));
    memcpy(grad,   t, sizeof(t));
    memcpy(mtum,   t, sizeof(t));
    
    return *this;
}

__BOTH__ Tensor&
Tensor::reshape(U64 sz) {
    if (sz == numel) {
        reset(data, numel, (t4_obj)ttype, grad_fn);   /// preserve ttype and fn
        MM_DB("  tensor#reshaped(%ld)\n", numel);
    }
    else {
        ERROR("  tensor#reshape sz != numel (%ld != %ld)\n", sz, numel);
    }
    return *this;
}

__BOTH__ Tensor&
Tensor::reshape(U32 h, U32 w) {
    const U16 s[4] = { 1, 1, 1, 1 };
    const U32 t[4] = { h, w, 1, 1 };
    U64 sz = (U64)h * w;
    if (sz == numel) {
        rank = 2;
        memcpy(stride, s, sizeof(s));
        memcpy(shape,  t, sizeof(t));
        MM_DB("  tensor#reshaped(%d,%d)\n", H(), W());
    }
    else {
        ERROR("  tensor#reshape sz != numel (%ld != %ld)\n", sz, numel);
    }
    return *this;
}

__BOTH__ Tensor&
Tensor::reshape(U32 n, U32 h, U32 w, U32 c) {
    const U16 s[4] = { 1, 1, 1, 1 };
    const U32 t[4] = { h, w, c, n };
    U64 sz = (U64)n * h * w * c;
    if (sz == numel) {
        rank = 4;
        memcpy(stride, s, sizeof(s));
        memcpy(shape,  t, sizeof(t));
        MM_DB("  tensor#reshaped(%d,%d,%d,%d)\n", N(), H(), W(), C());
    }
    else {
        ERROR("  tensor#reshape sz != numel (%ld != %ld)\n", sz, numel);
    }
    return *this;
}
__BOTH__ Tensor&
Tensor::reshape(U32 c1, U32 n, U32 h, U32 w, U32 c) {
    const U16 s[4] = { 1, 1, 1, 1 };
    const U32 t[4] = { h, w, c, n };
    U64 sz = (U64)c1 * n * h * w * c;
    if (sz == numel) {
        rank = 5;
        parm = c1;        /// use parm field, so we don't need s[5]
        memcpy(stride, s, sizeof(s));
        memcpy(shape,  t, sizeof(t));
        MM_DB("  tensor#reshaped(%d,%d,%d,%d,%d)\n", c1, N(), H(), W(), C());
    }
    else {
        ERROR("  tensor#reshape sz != numel (%ld != %ld)\n", sz, numel);
    }
    return *this;
}

__BOTH__ Tensor&
Tensor::identity() {
    const U32 W = this->W(), H = this->H(), C = this->C();
    for (U32 n = 0; n < N(); n++) {
        FORK3(k_identity, H, W, C, slice(n));
    }
    CDP_SYNC();
    return *this;
}

__BOTH__ Tensor&
Tensor::map(math_op op, DU v) {
    _OP(MATH_OP);
    MM_DB("  tensor#%s v=%g\n", _op[op], v);
    FORK1(k_math, numel, op, data, v);
    CDP_SYNC();
    return *this;
}

__BOTH__ Tensor&
Tensor::normalize(DU avg, DU std) {
    FORK1(k_ts_op, numel, SUB, data, avg, data);
    FORK1(k_ts_op, numel, DIV, data, std, data);
    CDP_SYNC();
    return *this;
}
///=======================================================================
/// Tensor debugger
///
__BOTH__ void
Tensor::_dump(DU *v, U32 H, U32 W, U32 C) {
    const U64 hw = H * W, sr = static_cast<U64>(sqrtf(hw));
    const U32 sh = (hw / sr) + ((hw - sr*sr) > 0L ? 1 : 0);
    const U32 h  = W > 1 ? H : (hw < 36L ? 1 : sh);
    const U32 w  = W > 1 ? W : (hw < 36L ? H : sr);
    
    DU *csum = new DU[C];
    for (U32 k = 0; k < C; k++) csum[k] = DU0;
    for (U32 i = 0; i < h; i++) {
        INFO("\n");
        DU sum = DU0;
        for (U32 k = 0; k < C; k++) {
            for (U32 j = 0; j < w; j++) {
                U64 n = j + i * w;
                if (n >= hw) { INFO(" ...."); continue; }
                
                DU  r = v[k + n * C];
                INFO("%5.2f", r);
                sum += r;
                csum[k] += r;
            }
            INFO("|");
        }
        INFO("Σ=%6.3f", sum);
    }
    if (h > 1) {
        INFO("\nΣΣ=");
        for (U32 k = 0; k < C; k++) INFO("%6.3f ", csum[k]);
    }
    delete csum;
}
///
///> _view - in ASCII art
///
__BOTH__ void
Tensor::_view(DU *v, U32 H, U32 W, U32 C, DU mean, DU scale) {
    auto map = [](DU v) {
        // static const char *lk = " .'`^\",:;Il!i><~+_-?][}{1)(|/tfjrxnuvczXYUJCLQ0OZmwqpdbkhao*#MW&8%B@$";                             // 69 shades
        static const char *lk = " .:-=+*#%@X";      // 11 shades
        //return lk[v < 10.0f ? (v < DU0 ? 10 : (int)v) : 9];
        int i = static_cast<int>((v + 1.0) * 5.5);
        return lk[i < 0 ? 0 : (i > 10 ? 10 : i)];
    };
    const U64 hw = H * W, sr = static_cast<U64>(sqrtf(hw));
    const U32 sh = (hw / sr) + ((hw - sr*sr) > 0L ? 1 : 0);
    const U32 w  = W > 1 ? W : (hw < 36L ? H : sr);
    const U32 h  = W > 1 ? H : (hw < 36L ? 1 : sh);

    DU *csum = new DU[C];
    for (U32 k = 0; k < C; k++) csum[k] = DU0;
    for (U32 i = 0; i < h; i++) {
        INFO("\n");
        for (U32 k = 0; k < C; k++) {
            for (U32 j = 0; j < w; j++) {
                U64 n = j + i * w;
                if (n >= hw) { INFO("  "); continue; }
                
                DU r0 = v[k + (j>0 ? n - 1 : n) * C];
                DU r1 = v[k + n * C];
                DU x0 = (r0 - mean) * scale;
                DU x1 = (((r0 + r1) * 0.5) - mean) * scale;

                INFO("%c%c", map(x0), map(x1));  // double width
                csum[k] += r1;
            }
            INFO("|");
        }
    }
    if (h > 1) {
        INFO("\nΣΣ=");
        for (U32 k = 0; k < C; k++) INFO("%6.3f ", csum[k]);
    }
    INFO("\n");
    
    delete csum;
}

__GPU__ void
Tensor::show(bool dump) {
    const U32 N  = this->N(), H = this->H(), W = this->W(), C = this->C();
    const U64 hw = (U64)H * W;

    DU mean  = avg();
    DU scale = 0.5 / std();            // P=95%
    for (U32 n = 0; n < N; n++) {
        DU *d = slice(n);
        if (dump || hw < 100) {
            INFO("\nn=%d", n);
            _dump(d, H, W, C);
        }
        if (hw > 36L) _view(d, H, W, C, mean, scale);
    }
    INFO("\n");
}

#endif // T4_DO_OBJ
