#include "hip/hip_runtime.h"
/** -*- c++ -*-
 * @file
 * @brief Model class - gradient descent functions implementation
 *
 * <pre>Copyright (C) 2022- GreenII, this file is distributed under BSD 3-Clause License.</pre>
 */
#include "model.h"

#if (T4_DO_OBJ && T4_DO_NN)
#include "dataset.h"

__KERN__ void k_sgd(
    DU *G, DU *DG, DU *M,                   ///< w, dw, and momemtum tensors
    U32 N, DU lr, DU b,                     ///< batch size, learn rate, beta(momemtum)
    U64 numel                               ///< HWC
    ) {
    for (U64 j = threadIdx.x; j < numel; j += blockDim.x) {
        if (ABS(b) < DU_EPS) G[j] -= lr * DG[j] / N;
        else {
            DU dg = DG[j] / N;                             ///< dG batch avg
            DU mi = M[j] = b * M[j] + (1.0 - b) * dg;      ///< momentum
            G[j] -= lr * mi;                               /// * update gradient
        }
        DG[j] = DU0;                                       /// * zero after batch
    }
}

__KERN__ void k_adam(
    DU *G, DU *DG, DU *M, DU *V,            ///< w, dw, and momemtum tensors
    U32 N, DU lrc, DU b1, DU b2,            ///< batch size,corrected learn rate, beta(momemtum)
    U64 numel                               ///< HWC
    ) {
    for (U64 j = threadIdx.x; j < numel; j += blockDim.x) {
        const DU dg = DG[j];                                     ///< dG (no batch avg)
        const DU mi = M[j] = b1 * M[j] + (DU1 - b1) * dg;        ///< momentum
        const DU vi = V[j] = b2 * V[j] + (DU1 - b2) * dg * dg;   ///< velocity
        
        G[j] -= lrc * mi / (SQRT(vi) + DU_EPS);                  /// * update gradient, clipped
        DG[j] = DU0;                                             /// * zero out dG for next round
    }
}

///
///> grad_alloc
///  @brief - allocate Momentum and Velocity tensors
///
#define M2X(i)     (in.mtum[i] ? _mmu->OBJ2X(*in.mtum[i]) : 0)
__GPU__ Model&
Model::grad_alloc(t4_optimizer op) {
    NN_DB("  #grad_alloc {\n");
    for (int i = 1; i < numel - 1; i++) {
        Tensor &in = (*this)[i];
        Tensor *w = in.grad[0], *dw = in.grad[2];   ///< filter tensor pointers
        Tensor *b = in.grad[1], *db = in.grad[3];   ///< bias tensor pointers

        bool do_w = dw && dw->is_same_shape(*w);    ///< exception: dropout
        bool do_b = db && db->is_same_shape(*b);    ///< exception: batchnorm
        
        switch (op) {
        case OPTI_SGD:
            in.mtum[0] = do_w ? w : NULL; in.mtum[2] = NULL;  /// * dummy
            in.mtum[1] = do_b ? b : NULL; in.mtum[3] = NULL;
            break;
        case OPTI_SGDM:
            if (do_w && !in.mtum[0]) {
                in.mtum[0] = &COPY(*dw).fill(DU0);  ///< m of w (zero filled)
                in.mtum[2] = NULL;                  ///< dummy
            }
            if (do_b && !in.mtum[1]) {
                in.mtum[1] = &COPY(*db).fill(DU0);  ///< m of b (zero filled)
                in.mtum[3] = NULL;                  ///< dummy
            }
            break;
        case OPTI_ADAM:
            if (do_w && !in.mtum[0]) {
                in.mtum[0] = &COPY(*dw).fill(DU0);  ///< m of w (zeor filled)
                in.mtum[2] = &COPY(*dw).fill(DU0);  ///< v of w (zero filled)
            }
            if (do_b && !in.mtum[1]) {
                in.mtum[1] = &COPY(*db).fill(DU0);  ///< m of b (zero filled)
                in.mtum[3] = &COPY(*db).fill(DU0);  ///< v of b (zero filled)
            }
            break;
        }
        NN_DB("    %d> %s do_w,b[%d,%d] mtum=%x,%x,%x,%x\n",
              i, d_nname(in.grad_fn), do_w, do_b,
              M2X(0), M2X(1), M2X(2), M2X(3));
    }
    NN_DB("  } #grad_alloc\n");
    return *this;
}
///
///> grandiant descent iterator
///
__GPU__ Model&
Model::gradient(const char *nm, t4_optimizer op, GdFunc fn, DU *parm) {
    auto step = [this, fn, parm](const char k,
        Tensor &g, Tensor &dg, Tensor &m, Tensor &v) {
        NN_DB("     %c[%2d,%2d,%2d,%2d] Σ=%6.3f - %6.3f",
              k, g.N(), g.H(), g.W(), g.C(), g.sum(), dg.sum());
#if MM_DEBUG        
        Tensor::_dump(g.data, g.H(), g.W(), g.C());
        Tensor::_dump(dg.data, dg.H(), dg.W(), dg.C());
        fn(parm, g, dg, m, v);                /// * execute grad function
        Tensor::_dump(g.data, g.H(), g.W(), g.C());
        Tensor::_dump(dg.data, dg.H(), dg.W(), dg.C());
#else  // !MM_DEBUG
        fn(parm, g, dg, m, v);                /// * execute grad function
#endif // MM_DEBUG
        DU sum0 = g.sum();
        NN_DB(" => %cΣ=%6.3f", k, sum0);
        if (max_norm > DU_EPS) {
            DU std = g.std();
            if (std > max_norm) {
                g *= max_norm / std;
                NN_DB(" std=%6.3% => adj. %cΣ=%6.3f");
            }
        }
        NN_DB("\n");
    };
    NLOG("\nModel::%s starts batch_sz=%d, lr=%7.4f, mtum/b1=%6.3f, b2=%6.3f max_norm=%6.3f {\n",
         nm, (*this)[1].N(), parm[0], parm[1], parm[2], max_norm);
    if (epoch==0 && _iter++==0) grad_alloc(op);   /// * allocate m & v tensors
    if (!train) return *this;                     /// * bail if not in trainning
    ///
    /// cascade execution layer by layer forward
    ///
    DU t0 = System::ms();                         ///< performance measurement
    for (int i = 1; i < numel - 1; i++) {         /// TODO: parallel layer update
        Tensor &in = (*this)[i];
        Tensor &w  = *in.grad[0], &dw = *in.grad[2];
        Tensor &b  = *in.grad[1], &db = *in.grad[3];

        if (*_trace) INFO("  %d> %s\n", i, d_nname(in.grad_fn));
        
        if (in.mtum[0]) step('w', w, dw, *in.mtum[0], *in.mtum[2]);
        if (in.mtum[1]) step('b', b, db, *in.mtum[1], *in.mtum[3]);
    }
    NLOG("} Model::%s %5.2f ms\n", nm, System::ms() - t0);
    return *this;
}
///
/// Stochastic Gradient Descent
/// Note: does not get affected by batch size
///       because filters are fixed size
///
__GPU__ Model&
Model::sgd(DU lr, DU b) {                          /// b=beta (momentum)
    auto update = [](DU parm[4], Tensor &g, Tensor &dg, Tensor &m, Tensor &v) {
        FORK1(k_sgd, g.numel, 
             g.data, dg.data, m.data,
             g.N(), parm[1], parm[2]);
        CDP_SYNC();
    };
    DU parm[3] = { lr, epoch ? b : DU0, DU0 };

    return gradient("sgd", ABS(b) < DU_EPS ? OPTI_SGDM : OPTI_SGDM, update, parm);
}

__GPU__ Model&
Model::adam(DU lr, DU b1, DU b2) {
    auto update = [](DU *parm, Tensor &g, Tensor &dg, Tensor &m, Tensor &v) {
        FORK1(k_adam, g.numel,
             g.data, dg.data, m.data, v.data,
             g.N(), parm[0], parm[1], parm[2]);
        CDP_SYNC();
    };
    DU parm[3] = {                    ///< learn rate, betas
        lr * SQRT(DU1 - POW(b2, epoch+1)) / (DU1 - POW(b1, epoch+1)),
        b1, b2
    };
    return gradient("adam", OPTI_ADAM, update, parm);
}
#endif  // (T4_DO_OBJ && T4_DO_NN)
//==========================================================================
