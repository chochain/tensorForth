#include "hip/hip_runtime.h"
/** -*- c++ -*-
 * @file
 * @brief Model class - backward propagation implementation
 *
 * <pre>Copyright (C) 2022- GreenII, this file is distributed under BSD 3-Clause License.</pre>
 */
#include "model.h"

#if (T4_DO_OBJ && T4_DO_NN)
///
/// convolution filter derivatives
/// TODO: stride, dilation, [C1]NCHW filter
///
template<int TS, int KS>    ///> tile size, kernel size
__KERN__ void k_dconv2d(
    DU *I, DU *F, DU *DF, DU *DB, DU *O,   ///< input I[HxW], F,DF[KSxKS], output O[HxW]
    int H, int W, int C0, bool train       ///< H1==H0, W1==W0, output Channels
    ) {
    __shared__ DU _I[T4_WARP_SQ];                    ///< input cache tile [16x16]
    __shared__ DU _O[T4_WARP_SQ];                    ///< output cache tile [16x16]

    const U32 KSQ= KS * KS;                          ///< save some muliplications
    const U32 tx = threadIdx.x, j1 = tx + blockIdx.x * TS;
    const U32 ty = threadIdx.y, i1 = ty + blockIdx.y * TS;
    const U32 c1 = blockIdx.z,  C1 = gridDim.z;      ///< channel deep
    const U64 z1 = ((U64)W * i1 + j1) * C1 + c1;     ///< input array index
    const U64 xy = (U64)T4_WARP_SZ * ty + tx;        ///< offset in cache window
    ///
    /// process z1, i.e. [TS, TS, C1] cells per kernel call
    ///
    const int i0 = i1 - INT(KS / 2);                 ///< dY coordinates
    const int j0 = j1 - INT(KS / 2);

    auto g = cg::this_thread_block();                ///< group all threads

    _I[xy] = (i1 < H && j1 < W) ? I[z1] : DU0;       ///< cached X (input) tile
    g.sync();

    for (U32 c0 = 0; c0 < C0; c0++) {                ///< each dY channel
        const U64 z0 = ((U64)W * i0 + j0) * C0 + c0; ///< output array index
        _O[xy] =                                     /// * cache dY (output) tile
            (i0 >= 0 && i0 < H && j0 >= 0 && j0 < W) /// * with zero padding
            ? O[z0] : DU0;                           /// * by channel
        g.sync();                                    /// * smem write barrier
        if (train && c1 == 0) {
            atomicAdd(&DB[c0], _O[xy]);              /// * dB += dY
        }
        const U64 zf = (U64)C0 * KSQ * c1 + c0;      ///< filter index F[C1,KS,KS,C0]
        if (tx < TS && ty < TS) {                    /// * within tile [12x12]
            DU *fx = &F[zf + (KSQ - 1) * C0];        ///< F[c1,KS-1,KS-1,c0] i.e. rot180
            DU *dfx= &DF[zf], *ox = &_O[xy];         ///< DF[c1,0,0,c0], dY
            DU sum = DU0;                            ///< dX sum (TSxTS threads)
            for (U32 y = 0; y < KS; y++) {           /// * process one KS * KS cell
                for (U32 x = 0; x < KS; x++) {
                    sum += (*fx) * ox[x];            /// * dX += F' @ dY (for each C1)
                    fx  -= C0;                       /// * walk F backward
                    if (!train) continue;
                    atomicAdd(dfx, ox[x] * _I[xy]);  /// * dF += dY * X (TSxTS threads)
                    dfx += C0;                       /// * DF[c1,0,1,c0]
                }
                ox += T4_WARP_SZ;
            }
            if (i1 < H && j1 < W) {                  /// * update input matrix
                if (c0 == 0) I[z1] = sum;            /// * update I (per C1)
                else         I[z1] += sum;
            }
        }
        g.sync();                                    /// * d read barrier
    }
}

__KERN__ void k_dlinear_dwdb(
    DU *I, DU *O, DU *DW, DU *DB,
    U64 HWC1, U64 HWC0, U32 C1, U32 C0
    ) {
    const U64 c1 = (U64)blockIdx.x * blockDim.x + threadIdx.x;
    const U64 c0 = (U64)blockIdx.y * blockDim.y + threadIdx.y; 
    const U64 cx = c0 * C1 + c1;
    const U32 n  = blockIdx.z;

    if (c0 < C0 && c1 < C1) {                          /// * TODO: shuffle-sum
        DU dy = O[HWC0 * n + c0];
        DU x  = I[HWC1 * n + c1];
        atomicAdd(&DW[cx], dy * x);                    /// * dw += dY @ X^t
        if (c1 == 0) atomicAdd(&DB[c0], dy);           /// * db += dY
    }
}

__KERN__ void k_dlinear_dx(
    DU *I, DU *O, DU *W,
    U64 HWC1, U64 HWC0, U32 C1, U32 C0
    ) {
    const U64 c1 = (U64)blockIdx.x * blockDim.x + threadIdx.x;
    const U64 c0 = (U64)blockIdx.y * blockDim.y + threadIdx.y;
    const U32 n  = blockIdx.z;
    const U64 cx = (U64)C1 * c0 + c1;

    if (c0 < C0 && c1 < C1) {                          /// * TODO: shuffle-sum
        DU dy = O[HWC0 * n + c0];
        DU *x = &I[HWC1 * n + c1];                     ///< pointer to X
        atomicAdd(x, W[cx] * dy);                      /// * dX = W^t * dY
    }
}

template<int KS>                                      /// kernel size
__KERN__ void k_dpool(
    t4_layer op,
    DU *I, DU *O,                                     ///< input, output buffers
    U32 H, U32 W                                      ///< output HW (C1==C0)
    ) {
    const U32 KSQ= KS * KS;
    const U64 HW = (U64)H * W;                        ///< HxW
    const U64 k0 = (U64)blockIdx.x * blockDim.x + threadIdx.x;
    const U32 j0 = k0 % W;                            ///< output x dim
    const U32 c  = blockIdx.y, C = gridDim.y;         ///< channel deep
    const U64 ns = HW * blockIdx.z * C;               ///< batch slice idx
    const U64 z0 = (U64)C * k0 + ns + c;              ///< output array index
    const U64 z1 = (U64)C * KS * j0 + KSQ * ((k0 - j0) * C + ns) + c;

    if (k0 < HW && c < C) {
        const U64 RI = (U64)KS * C * (W - 1);         ///< input cell row increment
        DU *ix = &I[z1], *t = ix;                     /// *ix input tensor cell
        DU2 v  = (op != L_AVGPOOL) ? *ix : O[z0] / KSQ;
        #pragma unroll
        for (U32 y = 0; y < KS; y++) {     /// * handle one kernel
            for (U32 x = 0; x < KS; x++) {
                DU dx = *ix;
                switch (op) {
                case L_AVGPOOL: *ix = v;             break;
                case L_MAXPOOL:
                    *ix = DU0;             /// * zero out all elements
                    if (dx > v) { v = dx; t = ix; }  break;
                case L_MINPOOL:
                    *ix = DU0;
                    if (dx < v) { v = dx; t = ix; }  break;
                case L_USAMPLE: *ix = O[z0];         break;
                }
                ix += C;                   /// * next cell
            }
            ix += RI;                      /// * next input row
        }
        if (op==L_MAXPOOL || op==L_MINPOOL) *t = O[z0];    /// * update arg cell
    }
}

__KERN__ void k_dactivate(
    DU *I, DU *F, DU *O,                   ///< input, filter, output
    U64 numel                              ///< tensor element count
    ) {
    const U64 j = (U64)blockIdx.x * blockDim.x + threadIdx.x;   ///< element index

    if (j < numel) I[j] = O[j] * F[j];     /// * Harmand product
}

__KERN__ void k_dbatchnorm_1(
    DU *I, DU *O, DU *X,                   ///< input, output, x_hat tensors
    DU *sum, DU *g_var,                    ///< sum(x_hat), gamma/(stdvar+e)
    U64 HW                                 ///< H0=H1, W0==W1 (C0==C1)
    ) {
    const U64 j  = (U64)threadIdx.x + blockIdx.x * blockDim.x;  ///< element index
    const U32 c  = blockIdx.y, C = gridDim.y;              ///< channel deep
    const U64 ns = HW * blockIdx.z * C;                    ///< batch slice index
    const U64 k  = (U64)C * j + ns + c;                    ///< output tensor index
    const DU  _N = 1.0 / gridDim.z;                        ///< 1.0/HWN

    if (j < HW) {
        I[k] = (O[k] - sum[c] * _N) * g_var[c];            /// * dX = g_var * (dout - sum(dout) / N)
        O[k] *= X[k];                                      /// * dout * x_hat
    }
}
__KERN__ void k_dbatchnorm_2(
    DU *I, DU *X, DU *sum,                 ///< input, x_hat
    U64 HW                                 ///< H0=H1, W0==W1 (C0==C1)
    ) {
    const U64 j  = (U64)blockIdx.x * blockDim.x + threadIdx.x;  ///< element index
    const U32 c  = blockIdx.y, C = gridDim.y;              ///< channel deep
    const U64 ns = HW * C * blockIdx.z;                    ///< batch slice index
    const U64 k  = (U64)C * j + ns + c;                    ///< output tensor index

    if (j < HW) I[k] -= X[k] * sum[c];
}
///
/// backprop: Neural Network back propegation
/// Note: cascade execution layer by layer backward
///
__GPU__ Model&
Model::broadcast(Tensor &tgt) {
    Tensor &out = (*this)[-1];                   ///< model output
    U64    HWC  = out.HWC();                     ///< sample size
    U32    N    = out.N();
    if (!_hot) _hot = &T4(N, HWC);               ///< allocate onehot vector if needed
    for (U32 n = 0; n < N; n++) {                /// * loop through batch, TODO: Kernel
        DU  v = tgt.data[n];                     ///< target vector
        DU *h = _hot->slice(n);                  ///< take a sample
        for (U64 i=0; i<HWC; i++) h[i] = v;      /// * broadcast [N,1] => [N,HWC]
    }
    return *this;
}

__GPU__ Model&
Model::backprop() {
    if (_hot) return backprop(*_hot);            /// * use default one-hot vector

    ERROR("Model#backprop missing onehot vector?\n");
    return *this;
}

__GPU__ Model&
Model::backprop(Tensor &tgt) {
    auto trace = [](DU t, int i, Tensor &in, Tensor &out) {
        printf("\n%6.2f:%2d> %s [%d,%d,%d,%d]\tp=%-2d <= out'Σ/n=%6.2f [%d,%d,%d,%d] ",
            t, i, d_nname(in.grad_fn),
            in.N(), in.H(), in.W(), in.C(), in.parm,
            out.sum() / out.N() / out.C(),
            out.N(), out.H(), out.W(), out.C());
    };
    if (_bloss(tgt)) return *this;                 /// * pre-calculate dLoss
    
    MM_DB("\nModel#backprop starts");
    DU  t0 = System::ms(), t1 = t0, tt;                   ///< performance measurement
    for (int i = numel - 2, j = 0; i > 0; i--, j++) {     /// numel=number of layers
        Tensor &in = (*this)[i], &out = (*this)[i + 1];
        if (_trace) {
            trace((tt=System::ms()) - t1, i, in, out); t1 = tt;
            _bstep(in, out);
            in.show();
        }
        else _bstep(in, out);
    }
    MM_DB("\nModel::backprop %5.2f ms\n", System::ms() - t0);
    return *this;
}
/// ========================================================================
/// private methods
///
__GPU__ int
Model::_bloss(Tensor &tgt) {                     ///> pre-calc dLoss
    Tensor &out = (*this)[-1];                   ///< output layer, used as dLoss
    if (tgt.numel != out.numel) {                /// * check dimensions of target vector
        ERROR("\nERROR: Onehot wrong shape[%d,%d,%d,%d] != [%d,%d,%d,%d]\n",
              tgt.N(), tgt.H(), tgt.W(), tgt.C(),
              out.N(), out.H(), out.W(), out.C());
        return 1;
    }
    MM_DB("\nModel#backprop: input dimensions OK, calculate dLoss");
    t4_layer fn = (*this)[-2].grad_fn;           ///< final activation layer
    switch (fn) {
    case L_SIGMOID:                              /// * sigmoid + BCE
    case L_SOFTMAX:                              /// * softmax + CE
    case L_LOGSMAX: out -= tgt;  break;          /// * log-softmax + NLL
    default:        out  = tgt;  break;          /// * pre-calc dLoss (pass thru)
    }
    if (_trace) out.show();                      /// * display loss if trace on

    return 0;
}

__GPU__ void
Model::_bstep(Tensor &in, Tensor &out) {
    ///
    /// layer function dispatcher
    ///
    t4_layer fn = in.grad_fn;                       ///< layer function
    switch(fn) {
    case L_CONV:    _bconv(in, out);         break; /// * convolution
    case L_LINEAR:  _blinear(in, out);       break; /// * out = w @ in + b
    case L_FLATTEN: in = out;                break; /// * pass dY to X
    case L_RELU:
    case L_TANH:                                    /// * in = (1 - t^2)*out
    case L_SIGMOID:                                 /// * in = s*(1 - s)*out
    case L_SELU:
    case L_LEAKYRL:
    case L_ELU:
    case L_DROPOUT: _bactivate(in, out);     break; /// * in = msk * out
    case L_SOFTMAX:                                 /// * softmax + CrossEntropy (pass thru)
    case L_LOGSMAX: in = out;                break; /// * log-softmax + NLL (pass thru)
    case L_MAXPOOL:
    case L_AVGPOOL:
    case L_MINPOOL: _bpool(in, out, fn);     break;
    case L_BATCHNM: _bbatchnorm(in, out);    break;
    case L_USAMPLE: _bupsample(in, out, fn); break;
    default: ERROR("Model#backprop layer=%d not supported\n", fn);
    }
}

#define TILE1    (T4_WARP_SZ)              /** 16, 1x1 conv */
#define TILE3    (T4_WARP_SZ - 3 + 1)      /** 14, 3x3 conv */
#define TILE5    (T4_WARP_SZ - 5 + 1)      /** 12, 5x5 conv */

__GPU__ int
Model::_bconv(Tensor &in, Tensor &out) {
    Tensor &w = *in.grad[0], &dw = *in.grad[2];      ///< filter tensor
    Tensor &b = *in.grad[1], &db = *in.grad[3];      ///< bias tensor

    MM_DB(" f[%d,%d,%d,%d], b[%ld]", w.N(), w.H(), w.W(), w.C(), b.numel);

    const U32 N = in.N(), H = in.H(), W = in.W();    ///< input dimensions
    const U32 C1 = in.C(), C0 = out.C();

    dim3 blk(T4_WARP_SZ, T4_WARP_SZ, 1);
    dim3 g1((W + TILE1 - 1) / TILE1, (H + TILE1 - 1) / TILE1, C1);
    dim3 g3((W + TILE3 - 1) / TILE3, (H + TILE3 - 1) / TILE3, C1);
    dim3 g5((W + TILE5 - 1) / TILE5, (H + TILE5 - 1) / TILE5, C1);

    for (U32 n = 0; n < N; n++) {                   ///< accumulative over N samples
        DU *d1 = in.slice(n), *d0 = out.slice(n);
        const U32 ks = w.H();                       ///< kernel size
        switch (ks) {
        case 1: k_dconv2d<TILE1,1><<<g1,blk,0,cudaStreamTailLaunch>>>(
                    d1, w.data, dw.data, db.data, d0, H, W, C0, train); break;
        case 3: k_dconv2d<TILE3,3><<<g3,blk,0,cudaStreamTailLaunch>>>(
                    d1, w.data, dw.data, db.data, d0, H, W, C0, train); break;
        case 5: k_dconv2d<TILE5,5><<<g5,blk,0,cudaStreamTailLaunch>>>(
                    d1, w.data, dw.data, db.data, d0, H, W, C0, train); break;
        default:
            ERROR("model_back#conv kernel_size %d not supported\n", ks);
            return -1;
        }
        // GPU_SYNC();
    }
    if (_trace > 1) _dump_dbdf(db, dw);
    return 0;
}

__GPU__ int
Model::_blinear(Tensor &in, Tensor &out) {
    auto qa_calc = [&in, &out](Tensor &w, Tensor &dw, Tensor &db, bool train) {
        const U32 N = in.N(), C1 = w.W(), C0 = w.H(); /// * weight dimensions
        for (U32 n = 0; n < N; n++) {               ///< acc over N samples
            DU *x = in.slice(n), *y = out.slice(n);
            if (train) {
                DU *dp = dw.data;
                for (U32 c0 = 0; c0 < C0; c0++) {   /// W[C0,C1]
                    DU yi = y[c0];
                    db[c0] += yi;                   /// * db += dY
                    for (U32 c1 =0; c1 < C1; c1++) {
                        *dp++ += yi * x[c1];        /// * dw += dY @ X^t
                    }
                }
            }
            DU *wd = w.data;
            for (U32 c1 = 0; c1 < C1; c1++) {       /// * dX = w^t @ dY
                DU sum = DU0;
                for (U32 c0 = 0; c0 < C0; c0++) {
                    sum += wd[c1 + c0 * C1] * y[c0];
                }
                x[c1] = sum;
            }
        }
    };                    
    Tensor &w  = *in.grad[0];                       ///< weight tensor
    Tensor &dw = *in.grad[2];                       ///< d_weight tensor
    Tensor &db = *in.grad[3];                       ///< d_bias tensor

    const U32 N  = out.N();                         ///< batch size (N1 == N0)
    const U32 C0 = w.H(), C1 = w.W();               ///< weight tensor dimensions
    const U64 E1 = in.HWC(), E0 = out.HWC();        ///< input, output element count

    MM_DB("\n\tdw[%d,%d] += out'[%ld,1] @ in^t[1,%ld]", C0, C1, E0, E1);
    MM_DB("\n\tin[%ld, 1] = w^t[%d,%d] @ out'[%ld,1]", E1, C1, C0, E0);

    if (w.numel < T4_WARP_SQ) {                     /// * threshold control
        MM_DB("*");
        qa_calc(w, dw, db, train);                  /// * serial mode (validation)
    }
    else {
        if (train) {
            FORK3(k_dlinear_dwdb, C1, C0, N,        /// * update dB, dW
                  in.data, out.data,
                  dw.data, db.data, E1, E0);
            // GPU_SYNC();
        }
        /// barrier for X (because we did N samples in one grid)
        in.map(FILL, DU0);                          /// * zero out dX
        FORK3(k_dlinear_dx, C1, C0, N,              /// * update dX
              in.data, out.data, w.data, E1, E0);
    }
    if (train && _trace > 1) {
         _dump_db(db);
         _dump_dw(dw, true);
    }
    return 0;
}

__GPU__ int
Model::_bactivate(Tensor &in, Tensor &out) {
    Tensor::ten_op(MUL, out, *in.grad[0], in);     /// * in = msk * out
    return 0;
}

__GPU__ int
Model::_bpool(Tensor &in, Tensor &out, t4_layer fn) {
    const U32 W = out.W(), H = out.H();           ///< output dimensions
    const U32 C = out.C(), N = out.N();
    const int ks = in.parm;                       ///< kernel size
    switch(ks) {
    case 2: FORK4(k_dpool<2>, fn, in.data, out.data, H, W); break;
    case 3: FORK4(k_dpool<3>, fn, in.data, out.data, H, W); break;
    default:
        ERROR("model#pooling kernel_size=%d not supported\n", ks);
        return -1;
    }
    // GPU_SYNC();
    return 0;
}
///
///> upsampling =~ reverse pooling (calls forward k_pool)
///
template<int KS>                                        /// forward declare (in forward.cu)
__KERN__ void k_pool(t4_layer op, DU *I, DU *O, U32 H, U32 W);
__GPU__ int
Model::_bupsample(Tensor &in, Tensor &out, t4_layer fn) {
    const U32 W  = in.W(), H = in.H();                  ///< input dimensions (reversed pool)
    const U32 C  = in.C(), N = in.N();
    const int me = (in.parm >> 8);                      ///< upsample method, TODO
    const int ks = (in.parm & 0xff);                    ///< kernel size

    switch(ks) {                                        /// by kernel size
    case 2: FORK4(k_pool<2>, fn, out.data, in.data, H, W); break;
    case 3: FORK4(k_pool<3>, fn, out.data, in.data, H, W); break;
    default:
        ERROR("model#upsample size=%d not supported\n", ks);
        return -1;
    }
    // GPU_SYNC();
    return 0;
}
///
///> batchnorm
///  @brief:
///    see https://kevinzakka.github.io/2016/09/14/batch_normalization/
///  @note
///    my own implmentation having dbeta and dgamma divided by HW
///    which is different from original document by does better
///    in preventing gradient explosion
///
extern __KERN__ void k_sum(DU *I, DU *sum, U64 HW);
__GPU__ int
Model::_bbatchnorm(Tensor &in, Tensor &out) {
    const U32 C = out.C(), N = out.N(), W = out.W(), H = out.H();   ///< C0==C1, N1=N0
    const U64 HW = (U64)W * H;

    DU *w   = &in.grad[0]->data[0];                    ///< weight/gamma (scale)
    DU *dw  = &in.grad[2]->data[0];                    ///< d_gamma
    DU *db  = &in.grad[2]->data[C];                    ///< d_beta
    DU *sum = &in.grad[1]->data[0];                    ///< batch sum
    DU *var = &in.grad[1]->data[C];                    ///< batch 1.0 / (var+e)^0.5
    DU *xht = in.grad[3]->data;                        ///< x_hat

    for (U32 c=0; c < C; c++) sum[c] = DU0;            /// * zero
    FORK4(k_sum, out.data, sum, HW);                   /// * capture out sum(dout)     
    // GPU_SYNC();
    
    for (U32 c=0; c < C; c++) {
        if (train) db[c] += (sum[c] /= HW);            /// * collect dbeta = sum(dout) (/ HW?)
        var[c] *= w[c];                                /// * var <= gamma * ivar
    }
    FORK4(k_dbatchnorm_1,                              /// * dX = gamma*ivar*(dout - sum(dout)/N)
        in.data, out.data, xht, sum, var, HW);         /// * also, dout *= x_hat
    // GPU_SYNC();
    
    for (U32 c=0; c < C; c++) sum[c] = DU0;            /// * zero
    FORK4(k_sum, out.data, sum, HW);                   /// * capture sum(dout * x_hat)
    // GPU_SYNC();

    for (U32 c=0; c < C; c++) {
        if (train) dw[c]  += (sum[c] /= HW);           /// * collect dgamma = sum(dout * x_hat)( / HW?)
        sum[c] *= var[c] / N;                          /// * scale sum
    }
    FORK4(k_dbatchnorm_2, in.data, xht, sum, HW);      /// * dX -= gamma*ivar*x_hat*sum(dout * x_hat) / N
    // GPU_SYNC();
    
    return 0;
}

#endif  // (T4_DO_OBJ && T4_DO_NN)
//==========================================================================
